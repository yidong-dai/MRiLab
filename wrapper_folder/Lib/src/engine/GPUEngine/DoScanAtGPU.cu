#include "hip/hip_runtime.h"


/************************************************************************
 MEX code for spin discrete evolution using IPP or Framewave and 
 parallel GPU computation (CUDA) written by Fang Liu (leoliuf@gmail.com).
************************************************************************/

/* system header */
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <vector>
/* MEX header */
#include <mex.h> 
#include "matrix.h"
/* nVIDIA CUDA header */
#include <hip/hip_runtime.h> 
/* OpenMP header*/
#include <omp.h>
/* Intel IPP header */
#ifdef IPP
#include <ipp.h>
#endif
/* AMD Framewave header */
#ifdef FW
#include <fwSignal.h>
#include <fwBase.h>
#define Ipp32f                  Fw32f
#define ippAlgHintFast          fwAlgHintFast
#define ippsMalloc_32f          fwsMalloc_32f
#define ippsFree                fwsFree
#define ippsZero_32f            fwsZero_32f
#define ippsZero_64f            fwsZero_64f
#define ippsSum_32f             fwsSum_32f
#define ippsCopy_32f            fwsCopy_32f
#define ippsAddC_32f            fwsAddC_32f
#define ippsAddC_32f_I          fwsAddC_32f_I
#define ippsAdd_32f             fwsAdd_32f 
#define ippsAdd_32f_I           fwsAdd_32f_I
#define ippsMulC_32f            fwsMulC_32f
#define ippsMulC_32f_I          fwsMulC_32f_I
#define ippsMul_32f             fwsMul_32f
#define ippsMul_32f_I           fwsMul_32f_I
#define ippsDiv_32f             fwsDiv_32f
#define ippsDivC_32f            fwsDivC_32f
#define ippsInv_32f_A24         fwsInv_32f_A24
#define ippsThreshold_LT_32f_I  fwsThreshold_LT_32f_I
#define ippsExp_32f_I           fwsExp_32f_I
#define ippsArctan_32f          fwsArctan_32f
#define ippsSqr_32f             fwsSqr_32f
#define ippsSqr_32f_I           fwsSqr_32f_I
#define ippsSqrt_32f_I          fwsSqrt_32f_I
#define ippsSin_32f_A24         fwsSin_32f_A24
#define ippsCos_32f_A24         fwsCos_32f_A24
#define ippsPolarToCart_32f     fwsPolarToCart_32f
#define ippsCartToPolar_32f     fwsCartToPolar_32f
#endif

/* for fixing error : identifier "IUnknown" is undefined" */
#ifdef _WIN32
#define WIN32_LEAN_AND_MEAN
#endif

#if defined(_WIN32) || defined(_WIN64)
#include <windows.h>
#endif

#define PI      3.14159265359 /* pi constant */

/* includes CUDA kernel */
#include "BlochKernel.cuh"
extern "C" bool mxUnshareArray(mxArray *array_ptr, bool noDeepCopy);

/* MEX entry function */
void mexFunction(int nlhs, mxArray *plhs[],int nrhs, const mxArray *prhs[])

{
/* pointers for VObj */
    double *Gyro;
    int SpinMxNum, SpinMxColNum, SpinMxRowNum, SpinMxSliceNum, SpinMxDimNum;
    const mwSize *SpinMxDims;
	float *Mz, *My, *Mx, *Rho, *T1, *T2;

/* pointers for VMag */
    float *dB0, *dWRnd, *Gzgrid, *Gygrid, *Gxgrid;
    
/* pointers for VCoi */
    float *RxCoilx, *RxCoily, *TxCoilmg, *TxCoilpe;
	double *RxCoilDefault, *TxCoilDefault;
    
/* pointers for VCtl */
    double *CS;
    int *TRNum, *MaxThreadNum, ThreadNum;
	int *ActiveThreadNum;
	int *GPUIndex;
    
/* pointers for VSeq */
    double *utsLine, *tsLine, *rfAmpLine, *rfPhaseLine, *rfFreqLine, *rfCoilLine, *GzAmpLine, *GyAmpLine, *GxAmpLine, *ADCLine, *ExtLine, *flagsLine;

/* pointers for VVar */
    double *t, *dt, *rfAmp, *rfPhase, *rfFreq, *rfCoil, *rfRef, *GzAmp, *GyAmp, *GxAmp, *ADC, *Ext, *KzTmp, *KyTmp, *KxTmp, *gpuFetch;
    int *utsi, *rfi, *Gzi, *Gyi, *Gxi, *ADCi, *Exti, *TRCount;
    
/* pointers for VSig */
    double *Sx, *Sy, *Kz, *Ky, *Kx;
	double *p_Sx, *p_Sy;
	
/* loop control */
    int i=0, j=0, s=0, Signali=0, Signalptr=0, PreSignalLen=0, SignalLen=0, SBufferLen=0, Typei, RxCoili, TxCoili;
    int MaxStep, MaxutsStep, MaxrfStep, MaxGzStep, MaxGyStep, MaxGxStep, *SpinNum, *TypeNum, *TxCoilNum, *RxCoilNum, *SignalNum;
    double flag[6];
    
/* IPP or FW buffer */
    Ipp32f buffer, *Sxbuffer, *Sybuffer;
	
/* function status */
    int ExtCall;
    
/* GPU execution sequence */
	std::vector<float> g_Sig;	

/* force breaking Copy-on-Write in Matlab */   
    mxUnshareArray(const_cast<mxArray *>(mexGetVariablePtr("global", "VObj")), true);
    mxUnshareArray(const_cast<mxArray *>(mexGetVariablePtr("global", "VMag")), true);
    mxUnshareArray(const_cast<mxArray *>(mexGetVariablePtr("global", "VCoi")), true);
    mxUnshareArray(const_cast<mxArray *>(mexGetVariablePtr("global", "VCtl")), true);
    mxUnshareArray(const_cast<mxArray *>(mexGetVariablePtr("global", "VSeq")), true);
	mxUnshareArray(const_cast<mxArray *>(mexGetVariablePtr("global", "VVar")), true);
    mxUnshareArray(const_cast<mxArray *>(mexGetVariablePtr("global", "VSig")), true);
    
/* assign pointers */
   Gyro            = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VObj"), 0, "Gyro"));
    Mz              = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VObj"), 0, "Mz"));
    My              = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VObj"), 0, "My"));
    Mx              = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VObj"), 0, "Mx"));
    Rho             = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VObj"), 0, "Rho"));
    T1              = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VObj"), 0, "T1"));
    T2              = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VObj"), 0, "T2"));
    SpinNum         = (int*)    mxGetData(mxGetField(mexGetVariablePtr("global", "VObj"), 0, "SpinNum"));
    TypeNum         = (int*)    mxGetData(mxGetField(mexGetVariablePtr("global", "VObj"), 0, "TypeNum"));
    
    dB0             = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VMag"), 0, "dB0"));
    dWRnd           = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VMag"), 0, "dWRnd"));
    Gzgrid          = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VMag"), 0, "Gzgrid"));
    Gygrid          = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VMag"), 0, "Gygrid"));
    Gxgrid          = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VMag"), 0, "Gxgrid"));
    
    TxCoilmg        = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VCoi"), 0, "TxCoilmg"));
    TxCoilpe        = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VCoi"), 0, "TxCoilpe"));
    RxCoilx         = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VCoi"), 0, "RxCoilx"));
    RxCoily         = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VCoi"), 0, "RxCoily"));
    TxCoilNum       = (int*)    mxGetData(mxGetField(mexGetVariablePtr("global", "VCoi"), 0, "TxCoilNum"));
    RxCoilNum       = (int*)    mxGetData(mxGetField(mexGetVariablePtr("global", "VCoi"), 0, "RxCoilNum"));
	TxCoilDefault   = (double*)    mxGetData(mxGetField(mexGetVariablePtr("global", "VCoi"), 0, "TxCoilDefault"));
    RxCoilDefault   = (double*)    mxGetData(mxGetField(mexGetVariablePtr("global", "VCoi"), 0, "RxCoilDefault"));
     
    CS              = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VCtl"), 0, "CS"));
    TRNum  			= (int*)    mxGetData(mxGetField(mexGetVariablePtr("global", "VCtl"), 0, "TRNum"));
    MaxThreadNum    = (int*)    mxGetData(mxGetField(mexGetVariablePtr("global", "VCtl"), 0, "MaxThreadNum"));
	ActiveThreadNum = (int*)    mxGetData(mxGetField(mexGetVariablePtr("global", "VCtl"), 0, "ActiveThreadNum"));
	GPUIndex		= (int*)    mxGetData(mxGetField(mexGetVariablePtr("global", "VCtl"), 0, "GPUIndex"));
    
    utsLine         = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VSeq"), 0, "utsLine"));
    tsLine          = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VSeq"), 0, "tsLine"));
    rfAmpLine       = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VSeq"), 0, "rfAmpLine"));
    rfPhaseLine     = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VSeq"), 0, "rfPhaseLine"));
    rfFreqLine      = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VSeq"), 0, "rfFreqLine"));
    rfCoilLine      = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VSeq"), 0, "rfCoilLine"));
    GzAmpLine       = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VSeq"), 0, "GzAmpLine"));
    GyAmpLine       = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VSeq"), 0, "GyAmpLine"));
    GxAmpLine       = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VSeq"), 0, "GxAmpLine"));
    ADCLine         = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VSeq"), 0, "ADCLine"));
    ExtLine         = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VSeq"), 0, "ExtLine"));
    flagsLine       = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VSeq"), 0, "flagsLine"));
    
    MaxStep         = mxGetNumberOfElements(mxGetField(mexGetVariablePtr("global", "VSeq"), 0, "tsLine"));
    MaxutsStep      = mxGetNumberOfElements(mxGetField(mexGetVariablePtr("global", "VSeq"), 0, "utsLine"));
    MaxrfStep       = mxGetNumberOfElements(mxGetField(mexGetVariablePtr("global", "VSeq"), 0, "rfAmpLine"));
    MaxGzStep       = mxGetNumberOfElements(mxGetField(mexGetVariablePtr("global", "VSeq"), 0, "GzAmpLine"));
    MaxGyStep       = mxGetNumberOfElements(mxGetField(mexGetVariablePtr("global", "VSeq"), 0, "GyAmpLine"));
    MaxGxStep       = mxGetNumberOfElements(mxGetField(mexGetVariablePtr("global", "VSeq"), 0, "GxAmpLine"));
	
	t               = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "t"));
    dt              = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "dt"));
    rfAmp           = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "rfAmp"));
    rfPhase         = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "rfPhase"));
    rfFreq          = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "rfFreq"));
    rfCoil          = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "rfCoil"));
    rfRef           = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "rfRef"));
    GzAmp           = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "GzAmp"));
    GyAmp           = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "GyAmp"));
    GxAmp           = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "GxAmp"));
    ADC             = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "ADC"));
    Ext             = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "Ext"));
    KzTmp           = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "Kz"));
    KyTmp           = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "Ky"));
    KxTmp           = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "Kx"));
	gpuFetch     	= (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "gpuFetch"));
    utsi            = (int*)    mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "utsi"));
    rfi             = (int*)    mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "rfi"));
    Gzi             = (int*)    mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "Gzi"));
    Gyi             = (int*)    mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "Gyi"));
    Gxi             = (int*)    mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "Gxi"));
    ADCi            = (int*)	mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "ADCi"));
    Exti            = (int*)    mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "Exti"));
    TRCount         = (int*)    mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "TRCount"));
	
	Sy              = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VSig"), 0, "Sy"));
    Sx              = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VSig"), 0, "Sx"));
    Kz              = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VSig"), 0, "Kz"));
    Ky              = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VSig"), 0, "Ky"));
    Kx              = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VSig"), 0, "Kx"));
    SignalNum       = (int*)    mxGetData(mxGetField(mexGetVariablePtr("global", "VSig"), 0, "SignalNum"));
    
/* get size of spin matrix */
    SpinMxDimNum    		= mxGetNumberOfDimensions(mxGetField(mexGetVariablePtr("global", "VObj"), 0, "Mz"));
    SpinMxDims      		= (mwSize*) mxCalloc(SpinMxDimNum, sizeof(mwSize));
    SpinMxDims      		= mxGetDimensions(mxGetField(mexGetVariablePtr("global", "VObj"), 0, "Mz"));
	
    SpinMxRowNum    		= SpinMxDims[0];
    SpinMxColNum    		= SpinMxDims[1];
    SpinMxNum       		= SpinMxDims[0] * SpinMxDims[1];
    if (SpinMxDimNum == 2){
        SpinMxSliceNum = 1;
    }else{
        SpinMxSliceNum = SpinMxDims[2];
    }
	
/* choose selected GPU */
	if( hipSuccess != hipSetDevice(*GPUIndex)){
        mexPrintf( "\n%s", hipGetErrorString(hipGetLastError()));
        return;
    }
	
/* set GPU grid & block configuration*/
    hipDeviceProp_t deviceProp;
    memset( &deviceProp, 0, sizeof(deviceProp));
    if( hipSuccess != hipGetDeviceProperties(&deviceProp, *GPUIndex)){
        mexPrintf( "\n%s", hipGetErrorString(hipGetLastError()));
        return;
    }

	dim3 dimGridImg(SpinMxColNum,1,1);
    dim3 dimBlockImg(1,SpinMxRowNum,1);

	for (i=SpinMxColNum - 1; i >= deviceProp.multiProcessorCount; i--){
		if ( SpinMxNum % i == 0 ){
			if (SpinMxNum/i > deviceProp.maxThreadsPerBlock) break;
			if ((SpinMxNum/i)*63 > deviceProp.regsPerBlock) break; // 63 registers per thread for current kernel
			dimGridImg.x = i;
		    dimBlockImg.y = SpinMxNum/i;
		}
	}
	i=0;
	
/* allocate device memory for matrices */
    float *d_Mz = NULL;
    hipMalloc( (void**) &d_Mz, SpinMxNum * SpinMxSliceNum * (*SpinNum) * (*TypeNum) * sizeof(float)) ;
	hipMemcpy( d_Mz, Mz, SpinMxNum * SpinMxSliceNum * (*SpinNum) * (*TypeNum) * sizeof(float), hipMemcpyHostToDevice ) ;
    
    float *d_My = NULL;
    hipMalloc( (void**) &d_My, SpinMxNum * SpinMxSliceNum * (*SpinNum) * (*TypeNum) * sizeof(float)) ;
	hipMemcpy( d_My, My, SpinMxNum * SpinMxSliceNum * (*SpinNum) * (*TypeNum) * sizeof(float), hipMemcpyHostToDevice ) ;
    
    float *d_Mx = NULL;
    hipMalloc( (void**) &d_Mx, SpinMxNum * SpinMxSliceNum * (*SpinNum) * (*TypeNum) * sizeof(float)) ;
	hipMemcpy( d_Mx, Mx, SpinMxNum * SpinMxSliceNum * (*SpinNum) * (*TypeNum) * sizeof(float), hipMemcpyHostToDevice ) ;
    
    float *d_dWRnd = NULL;
    hipMalloc( (void**) &d_dWRnd, SpinMxNum * SpinMxSliceNum * (*SpinNum) * (*TypeNum) * sizeof(float)) ;
	hipMemcpy( d_dWRnd, dWRnd, SpinMxNum * SpinMxSliceNum * (*SpinNum) * (*TypeNum) * sizeof(float), hipMemcpyHostToDevice ) ;
    
    float *d_Rho = NULL;
    hipMalloc( (void**) &d_Rho, SpinMxNum * SpinMxSliceNum * (*TypeNum) * sizeof(float)) ;
	hipMemcpy( d_Rho, Rho, SpinMxNum * SpinMxSliceNum * (*TypeNum) * sizeof(float), hipMemcpyHostToDevice ) ;
    
    float *d_T1 = NULL;
    hipMalloc( (void**) &d_T1, SpinMxNum * SpinMxSliceNum * (*TypeNum) * sizeof(float)) ;
	hipMemcpy( d_T1, T1, SpinMxNum * SpinMxSliceNum * (*TypeNum) * sizeof(float), hipMemcpyHostToDevice ) ;
    
    float *d_T2 = NULL;
    hipMalloc( (void**) &d_T2, SpinMxNum * SpinMxSliceNum * (*TypeNum) * sizeof(float)) ;
	hipMemcpy( d_T2, T2, SpinMxNum * SpinMxSliceNum * (*TypeNum) * sizeof(float), hipMemcpyHostToDevice ) ;
    
    float *d_Gzgrid = NULL;
    hipMalloc( (void**) &d_Gzgrid, SpinMxNum * SpinMxSliceNum * sizeof(float)) ;
	hipMemcpy( d_Gzgrid, Gzgrid, SpinMxNum * SpinMxSliceNum * sizeof(float), hipMemcpyHostToDevice ) ;
    
    float *d_Gygrid = NULL;
    hipMalloc( (void**) &d_Gygrid, SpinMxNum * SpinMxSliceNum * sizeof(float)) ;
	hipMemcpy( d_Gygrid, Gygrid, SpinMxNum * SpinMxSliceNum * sizeof(float), hipMemcpyHostToDevice ) ;
    
    float *d_Gxgrid = NULL;
    hipMalloc( (void**) &d_Gxgrid, SpinMxNum * SpinMxSliceNum * sizeof(float)) ;
	hipMemcpy( d_Gxgrid, Gxgrid, SpinMxNum * SpinMxSliceNum * sizeof(float), hipMemcpyHostToDevice ) ;
    
    float *d_dB0 = NULL;
    hipMalloc( (void**) &d_dB0, SpinMxNum * SpinMxSliceNum * sizeof(float)) ;
	hipMemcpy( d_dB0, dB0, SpinMxNum * SpinMxSliceNum * sizeof(float), hipMemcpyHostToDevice ) ;

    float *d_TxCoilmg = NULL;
    hipMalloc( (void**) &d_TxCoilmg, SpinMxNum * SpinMxSliceNum * (*TxCoilNum) * sizeof(float)) ;
	hipMemcpy( d_TxCoilmg, TxCoilmg, SpinMxNum * SpinMxSliceNum * (*TxCoilNum) * sizeof(float), hipMemcpyHostToDevice ) ;

    float *d_TxCoilpe = NULL;
    hipMalloc( (void**) &d_TxCoilpe, SpinMxNum * SpinMxSliceNum * (*TxCoilNum) * sizeof(float)) ;
	hipMemcpy( d_TxCoilpe, TxCoilpe, SpinMxNum * SpinMxSliceNum * (*TxCoilNum) * sizeof(float), hipMemcpyHostToDevice ) ;
	
	float *d_RxCoilx = NULL;
    hipMalloc( (void**) &d_RxCoilx, SpinMxNum * SpinMxSliceNum * (*RxCoilNum) * sizeof(float)) ;
	hipMemcpy( d_RxCoilx, RxCoilx, SpinMxNum * SpinMxSliceNum * (*RxCoilNum) * sizeof(float), hipMemcpyHostToDevice ) ;

	float *d_RxCoily = NULL;
    hipMalloc( (void**) &d_RxCoily, SpinMxNum * SpinMxSliceNum * (*RxCoilNum) * sizeof(float)) ;
	hipMemcpy( d_RxCoily, RxCoily, SpinMxNum * SpinMxSliceNum * (*RxCoilNum) * sizeof(float), hipMemcpyHostToDevice ) ;
	
    double *d_CS = NULL;
    hipMalloc( (void**) &d_CS, *TypeNum * sizeof(double)) ;
	hipMemcpy( d_CS, CS, *TypeNum * sizeof(double), hipMemcpyHostToDevice ) ;
	
/* allocate device memory for GPU execution sequence*/
    float *d_Sig = NULL;
    hipMalloc( (void**) &d_Sig, (5+3*(*TxCoilNum)) * MaxutsStep * sizeof(float)) ;
	
/* set CPU signal buffer */
	Sxbuffer    = ippsMalloc_32f(SpinMxNum * PreSignalLen * (*TypeNum) * (*RxCoilNum));
	Sybuffer    = ippsMalloc_32f(SpinMxNum * PreSignalLen * (*TypeNum) * (*RxCoilNum));

/* allocate device memory for buffering acquired signal */
    float *d_Sx = NULL;
    hipMalloc( (void**) &d_Sx, SpinMxNum * PreSignalLen * (*TypeNum) * (*RxCoilNum) * sizeof(float)) ;
    float *d_Sy = NULL;
    hipMalloc( (void**) &d_Sy, SpinMxNum * PreSignalLen * (*TypeNum) * (*RxCoilNum) * sizeof(float)) ;

/* start simulator execution loop */
	mexPrintf("------ Current active GPU device : %s ------\n", &deviceProp.name[0]);
    mexPrintf("TR Counts: %d of %d\n", 1, *TRNum);
    while (i < MaxStep){
        /* check MR sequence pulse flag */
        flag[0]=0;
        flag[1]=0;
        flag[2]=0;
        flag[3]=0;
        flag[4]=0;
        flag[5]=0;
        if (tsLine[i]!=tsLine[i+1]){
            flag[0]+=flagsLine[i*6];
            flag[1]+=flagsLine[i*6+1];
            flag[2]+=flagsLine[i*6+2];
            flag[3]+=flagsLine[i*6+3];
            flag[4]+=flagsLine[i*6+4];
            flag[5]+=flagsLine[i*6+5];
            i++;
        }
        else{
            flag[0]+=flagsLine[i*6];
            flag[1]+=flagsLine[i*6+1];
            flag[2]+=flagsLine[i*6+2];
            flag[3]+=flagsLine[i*6+3];
            flag[4]+=flagsLine[i*6+4];
            flag[5]+=flagsLine[i*6+5];
            
            while (tsLine[i]==tsLine[i+1]){
                flag[0]+=flagsLine[(i+1)*6];
                flag[1]+=flagsLine[(i+1)*6+1];
                flag[2]+=flagsLine[(i+1)*6+2];
                flag[3]+=flagsLine[(i+1)*6+3];
                flag[4]+=flagsLine[(i+1)*6+4];
                flag[5]+=flagsLine[(i+1)*6+5];
                i++;
                if (i==MaxStep-1){
                    break;
                }
            }
            i++;
        }
        
        /* update pulse status */
        *t 	= *(utsLine + *utsi);
        *dt 	= *(utsLine + (int)min(*utsi+1, MaxutsStep-1))-*(utsLine + *utsi);
        *utsi = (int)min(*utsi+1, MaxutsStep-1);
		if (*dt > 0) g_Sig.push_back((float)*dt);
		
        if (flag[0]>=1 ){ /* update rfAmp, rfPhase, rfFreq, rfCoil for multiple rf lines */
            for (j = 0; j < flag[0]; j++){
				 *rfCoil = *(rfCoilLine+ *rfi);
				 TxCoili = (int)(*rfCoil);
				 s = *rfi + 1;
				 while (s < MaxrfStep){
					if (*rfCoil == *(rfCoilLine + s)){
						if (fabs(*(rfAmpLine+ *rfi)) <= fabs(*(rfAmpLine + s)))
							*(rfAmp + TxCoili - 1)= *(rfAmpLine+ *rfi);
						else
							*(rfAmp + TxCoili - 1)= *(rfAmpLine+ s);
	                    
						if (fabs(*(rfPhaseLine+ *rfi)) <= fabs(*(rfPhaseLine + s)))
							*(rfPhase + TxCoili - 1)= *(rfPhaseLine+ *rfi);
						else
							*(rfPhase + TxCoili - 1)= *(rfPhaseLine+ s);
	                    
						if (fabs(*(rfFreqLine+ *rfi)) <= fabs(*(rfFreqLine + s)))
							*(rfFreq + TxCoili - 1)= *(rfFreqLine+ *rfi);
						else
							*(rfFreq + TxCoili - 1)= *(rfFreqLine+ s);
						break;
					}
					s++;
				 }
				 (*rfi)++;
            }
			
			for (j = 0; j < *TxCoilNum; j++){ /* multi-Tx, deal with rfPhase */
				if (rfAmp[j]<0){
					rfAmp[j]=fabs(rfAmp[j]);
					rfPhase[j]=rfPhase[j]+PI;
				}
			}
			
        }
		if (*dt > 0){
			for (j = 0; j < *TxCoilNum; j++){
				g_Sig.push_back((float)rfAmp[j]);
				g_Sig.push_back((float)rfPhase[j]);
				g_Sig.push_back((float)rfFreq[j]);
			}
		}

        if (flag[1]==1 ){ /* update GzAmp */
            if (fabs(*(GzAmpLine+ *Gzi)) <= fabs(*(GzAmpLine + (int)min(*Gzi+1, MaxGzStep-1))))
                *GzAmp = *(GzAmpLine+ *Gzi);
            else
                *GzAmp = *(GzAmpLine+ *Gzi+1);
            
            (*Gzi)++;
        }
        if (*dt > 0) g_Sig.push_back((float)*GzAmp);
		
        if (flag[2]==1 ){ /* update GyAmp */
            if (fabs(*(GyAmpLine+ *Gyi)) <= fabs(*(GyAmpLine + (int)min(*Gyi+1, MaxGyStep-1))))
                *GyAmp = *(GyAmpLine+ *Gyi);
            else
                *GyAmp = *(GyAmpLine+ *Gyi+1);
            
            (*Gyi)++;
        }
        if (*dt > 0) g_Sig.push_back((float)*GyAmp);
		
        if (flag[3]==1 ){ /* update GxAmp */
            if (fabs(*(GxAmpLine+ *Gxi)) <= fabs(*(GxAmpLine + (int)min(*Gxi+1, MaxGxStep-1))))
                *GxAmp = *(GxAmpLine+ *Gxi);
            else
                *GxAmp = *(GxAmpLine+ *Gxi+1);
            
            (*Gxi)++;
        }			
		if (*dt > 0) g_Sig.push_back((float)*GxAmp);
        
        *ADC = 0;   /* avoid ADC overflow */
        if (flag[4]==1){ /* update ADC */
            *ADC = *(ADCLine+ *ADCi);
            (*ADCi)++;
        }
		if (*dt > 0) g_Sig.push_back((float)*ADC);
		
		if (*ADC == 1){
			/* update k-space */
            Kz[Signali] += *KzTmp;
            Ky[Signali] += *KyTmp;
            Kx[Signali] += *KxTmp;
            Signali++;
		}
		
		 /* update Kz, Ky & Kx buffer */
        *KzTmp +=(*GzAmp)*(*dt)*(*Gyro/(2*PI));
        *KyTmp +=(*GyAmp)*(*dt)*(*Gyro/(2*PI));
        *KxTmp +=(*GxAmp)*(*dt)*(*Gyro/(2*PI));
		
        if (flag[5]==1){ /* update Ext */
            *Ext = *(ExtLine+ *Exti);
            /* execute extended process */
            if (*Ext != 0){
				if (g_Sig.size() !=0){
				
					/* calculate signal length */
					SignalLen = Signali-Signalptr;

					/* reset buffer */
					if (PreSignalLen!=SignalLen && SignalLen>0){
						PreSignalLen = SignalLen;
						/* allocate device memory for acquired signal buffer */
						hipFree(d_Sx);
						hipFree(d_Sy);
						hipMalloc( (void**) &d_Sx, SpinMxNum * SignalLen * (*TypeNum) * (*RxCoilNum) * sizeof(float)) ;
						hipMalloc( (void**) &d_Sy, SpinMxNum * SignalLen * (*TypeNum) * (*RxCoilNum) * sizeof(float)) ;
						/* zero signal buffer */
						hipMemset(d_Sx, 0 ,SpinMxNum * SignalLen * (*TypeNum) * (*RxCoilNum) * sizeof(float)); /* only work for 0 */
						hipMemset(d_Sy, 0 ,SpinMxNum * SignalLen * (*TypeNum) * (*RxCoilNum) * sizeof(float)); /* only work for 0 */
						/* set buffer */
						ippsFree(Sxbuffer);
						ippsFree(Sybuffer);
						Sxbuffer = ippsMalloc_32f(SpinMxNum * SignalLen * (*TypeNum) * (*RxCoilNum));
						Sybuffer = ippsMalloc_32f(SpinMxNum * SignalLen * (*TypeNum) * (*RxCoilNum));
					}

					/* avoid shared memory overflow */
					if (g_Sig.size() * sizeof(float) > deviceProp.sharedMemPerBlock){
						SBufferLen = 0;
					}else{
						SBufferLen = g_Sig.size() * sizeof(float);
					}

					/* upload GPU sequence */
					hipMemcpy( d_Sig, 	&g_Sig[0], 	g_Sig.size() * sizeof(float),	hipMemcpyHostToDevice ) ;

					/* call GPU kernel for spin discrete precessing */
					BlochKernelNormalGPU<<< dimGridImg, dimBlockImg, SBufferLen >>>
										((float)*Gyro, d_CS, d_Rho, d_T1, d_T2, d_Mz, d_My, d_Mx,
										d_dB0, d_dWRnd, d_Gzgrid, d_Gygrid, d_Gxgrid, d_TxCoilmg, d_TxCoilpe, d_RxCoilx, d_RxCoily,
										d_Sig, (float)*RxCoilDefault, (float)*TxCoilDefault,
										d_Sx, d_Sy, (float)*rfRef, SignalLen, SBufferLen,
										SpinMxColNum, SpinMxRowNum, SpinMxSliceNum, *SpinNum, *TypeNum, *TxCoilNum, *RxCoilNum, g_Sig.size()/(5+3*(*TxCoilNum)));
					hipDeviceSynchronize();
					g_Sig.clear();
					Signalptr = Signali; /* shift signal array pointer */
				}
				
				/* signal acquisition */
				if (SignalLen>0){
					/* get Sx, Sy buffer from GPU */
					hipMemcpy( Sybuffer, d_Sy, SpinMxNum * SignalLen * (*RxCoilNum) * (*TypeNum) * sizeof(float), hipMemcpyDeviceToHost ) ;
					hipMemcpy( Sxbuffer, d_Sx, SpinMxNum * SignalLen * (*RxCoilNum) * (*TypeNum) * sizeof(float), hipMemcpyDeviceToHost ) ;
					
					/* sum MR signal via openMP */
					for (Typei = 0; Typei < *TypeNum; Typei++){
						for (RxCoili = 0; RxCoili < *RxCoilNum; RxCoili++){  /* signal acquisition per Rx coil */
							#pragma omp parallel
							{   
								#pragma omp for private(j, s, p_Sx, p_Sy, buffer) 
								for (j=0; j < SignalLen; j++){
									
									if (j==0){
										*ActiveThreadNum = omp_get_num_threads();
									}
									
									s=Signali-SignalLen+j;
									p_Sx = Sx + (Typei*(*RxCoilNum)*(*SignalNum)+RxCoili*(*SignalNum)+s);
									p_Sy = Sy + (Typei*(*RxCoilNum)*(*SignalNum)+RxCoili*(*SignalNum)+s);
								
									ippsSum_32f(&Sxbuffer[Typei * (SpinMxNum * SignalLen * (*RxCoilNum)) + RxCoili * (SpinMxNum * SignalLen) +  j*SpinMxNum], SpinMxNum, &buffer, ippAlgHintFast);
									*p_Sx = (double)buffer;
									ippsSum_32f(&Sybuffer[Typei * (SpinMxNum * SignalLen * (*RxCoilNum)) + RxCoili * (SpinMxNum * SignalLen) +  j*SpinMxNum], SpinMxNum, &buffer, ippAlgHintFast);
									*p_Sy = (double)buffer;
								
								}
							}
						}       
					}
					
					/* zero signal buffer */
					hipMemset(d_Sx, 0 ,SpinMxNum * SignalLen * (*TypeNum) * (*RxCoilNum) * sizeof(float)); /* only work for 0 */
					hipMemset(d_Sy, 0 ,SpinMxNum * SignalLen * (*TypeNum) * (*RxCoilNum) * sizeof(float)); /* only work for 0 */
				}

			    /* fetch GPU data? */
                ExtCall = mexEvalString("DoGPUFetch");
                if (ExtCall){
                    mexErrMsgTxt("Extended process encounters ERROR!");
                    return;
                }
				
				if (*gpuFetch !=0){
					/* fetch data from GPU */
					hipMemcpy( Mz, d_Mz, SpinMxNum * SpinMxSliceNum * (*SpinNum) * (*TypeNum) * sizeof(float), hipMemcpyDeviceToHost );
					hipMemcpy( My, d_My, SpinMxNum * SpinMxSliceNum * (*SpinNum) * (*TypeNum) * sizeof(float), hipMemcpyDeviceToHost );
					hipMemcpy( Mx, d_Mx, SpinMxNum * SpinMxSliceNum * (*SpinNum) * (*TypeNum) * sizeof(float), hipMemcpyDeviceToHost );
					hipMemcpy( dWRnd, d_dWRnd, SpinMxNum * SpinMxSliceNum * (*SpinNum) * (*TypeNum) * sizeof(float), hipMemcpyDeviceToHost );
					hipMemcpy( Rho, d_Rho, SpinMxNum * SpinMxSliceNum * (*TypeNum) * sizeof(float), hipMemcpyDeviceToHost );
					hipMemcpy( T1, d_T1, SpinMxNum * SpinMxSliceNum * (*TypeNum) * sizeof(float), hipMemcpyDeviceToHost );
					hipMemcpy( T2, d_T2, SpinMxNum * SpinMxSliceNum * (*TypeNum) * sizeof(float), hipMemcpyDeviceToHost );
					hipMemcpy( Gzgrid, d_Gzgrid, SpinMxNum * SpinMxSliceNum * sizeof(float), hipMemcpyDeviceToHost );
					hipMemcpy( Gygrid, d_Gygrid, SpinMxNum * SpinMxSliceNum * sizeof(float), hipMemcpyDeviceToHost );
					hipMemcpy( Gxgrid, d_Gxgrid, SpinMxNum * SpinMxSliceNum * sizeof(float), hipMemcpyDeviceToHost );
					hipMemcpy( dB0, d_dB0, SpinMxNum * SpinMxSliceNum * sizeof(float), hipMemcpyDeviceToHost );
					hipMemcpy( TxCoilmg, d_TxCoilmg, SpinMxNum * SpinMxSliceNum * (*TxCoilNum) * sizeof(float), hipMemcpyDeviceToHost );
					hipMemcpy( TxCoilpe, d_TxCoilpe, SpinMxNum * SpinMxSliceNum * (*TxCoilNum) * sizeof(float), hipMemcpyDeviceToHost );
					hipMemcpy( RxCoilx, d_RxCoilx, SpinMxNum * SpinMxSliceNum * (*RxCoilNum) * sizeof(float), hipMemcpyDeviceToHost );
					hipMemcpy( RxCoily, d_RxCoily, SpinMxNum * SpinMxSliceNum * (*RxCoilNum) * sizeof(float), hipMemcpyDeviceToHost );
				}

                /* execute extended process */
                ExtCall = mexEvalString("DoExtPlugin");
                if (ExtCall){
                    mexErrMsgTxt("Extended process encounters ERROR!");
                    return;
                }
				
                /* update pointers, avoid pointer change between Matlab and Mex call */
                t               = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "t"));
                dt              = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "dt"));
                rfAmp           = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "rfAmp"));
                rfPhase         = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "rfPhase"));
                rfFreq          = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "rfFreq"));
                rfCoil          = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "rfCoil"));
                rfRef           = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "rfRef"));
                GzAmp           = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "GzAmp"));
                GyAmp           = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "GyAmp"));
                GxAmp           = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "GxAmp"));
                ADC             = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "ADC"));
                Ext             = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "Ext"));
                KzTmp           = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "Kz"));
                KyTmp           = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "Ky"));
                KxTmp           = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "Kx"));
                gpuFetch     	= (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "gpuFetch"));
                utsi            = (int*)    mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "utsi"));
                rfi             = (int*)    mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "rfi"));
                Gzi             = (int*)    mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "Gzi"));
                Gyi             = (int*)    mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "Gyi"));
                Gxi             = (int*)    mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "Gxi"));
                ADCi            = (int*)	mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "ADCi"));
                Exti            = (int*)    mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "Exti"));
                TRCount         = (int*)    mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "TRCount"));

				if (*gpuFetch !=0){
					*gpuFetch =0;
					/* update pointers, avoid pointer change between Matlab and Mex call */
					Mz          = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VObj"), 0, "Mz"));
					My          = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VObj"), 0, "My"));
					Mx          = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VObj"), 0, "Mx"));
					Rho         = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VObj"), 0, "Rho"));
					T1          = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VObj"), 0, "T1"));
					T2          = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VObj"), 0, "T2"));
					dWRnd       = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VMag"), 0, "dWRnd"));
					dB0         = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VMag"), 0, "dB0"));
					Gzgrid      = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VMag"), 0, "Gzgrid"));
					Gygrid      = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VMag"), 0, "Gygrid"));
					Gxgrid      = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VMag"), 0, "Gxgrid"));
					TxCoilmg    = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VCoi"), 0, "TxCoilmg"));
					TxCoilpe    = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VCoi"), 0, "TxCoilpe"));
					RxCoilx     = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VCoi"), 0, "RxCoilx"));
					RxCoily     = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VCoi"), 0, "RxCoily"));

					/* send data back to GPU */
					hipMemcpy( d_Mz, Mz, SpinMxNum * SpinMxSliceNum * (*SpinNum) * (*TypeNum) * sizeof(float), hipMemcpyHostToDevice );
					hipMemcpy( d_My, My, SpinMxNum * SpinMxSliceNum * (*SpinNum) * (*TypeNum) * sizeof(float), hipMemcpyHostToDevice );
					hipMemcpy( d_Mx, Mx, SpinMxNum * SpinMxSliceNum * (*SpinNum) * (*TypeNum) * sizeof(float), hipMemcpyHostToDevice );
					hipMemcpy( d_dWRnd, dWRnd, SpinMxNum * SpinMxSliceNum * (*SpinNum) * (*TypeNum) * sizeof(float), hipMemcpyHostToDevice );
					hipMemcpy( d_Rho, Rho, SpinMxNum * SpinMxSliceNum * (*TypeNum) * sizeof(float), hipMemcpyHostToDevice );
					hipMemcpy( d_T1, T1, SpinMxNum * SpinMxSliceNum * (*TypeNum) * sizeof(float), hipMemcpyHostToDevice );
					hipMemcpy( d_T2, T2, SpinMxNum * SpinMxSliceNum * (*TypeNum) * sizeof(float), hipMemcpyHostToDevice );
					hipMemcpy( d_Gzgrid, Gzgrid, SpinMxNum * SpinMxSliceNum * sizeof(float), hipMemcpyHostToDevice );
					hipMemcpy( d_Gygrid, Gygrid, SpinMxNum * SpinMxSliceNum * sizeof(float), hipMemcpyHostToDevice );
					hipMemcpy( d_Gxgrid, Gxgrid, SpinMxNum * SpinMxSliceNum * sizeof(float), hipMemcpyHostToDevice );
					hipMemcpy( d_dB0, dB0, SpinMxNum * SpinMxSliceNum * sizeof(float), hipMemcpyHostToDevice );
					hipMemcpy( d_TxCoilmg, TxCoilmg, SpinMxNum * SpinMxSliceNum * (*TxCoilNum) * sizeof(float), hipMemcpyHostToDevice );
					hipMemcpy( d_TxCoilpe, TxCoilpe, SpinMxNum * SpinMxSliceNum * (*TxCoilNum) * sizeof(float), hipMemcpyHostToDevice );
					hipMemcpy( d_RxCoilx, RxCoilx, SpinMxNum * SpinMxSliceNum * (*RxCoilNum) * sizeof(float), hipMemcpyHostToDevice );
					hipMemcpy( d_RxCoily, RxCoily, SpinMxNum * SpinMxSliceNum * (*RxCoilNum) * sizeof(float), hipMemcpyHostToDevice );
				}
            }
            (*Exti)++;
        }
        
        if (flag[0]+flag[1]+flag[2]+flag[3]+flag[4]+flag[5] == 0){ /* reset VVar */
            ippsZero_64f(rfAmp, *TxCoilNum);
            ippsZero_64f(rfPhase, *TxCoilNum);
            ippsZero_64f(rfFreq, *TxCoilNum);
            *GzAmp = 0;
            *GyAmp = 0;
            *GxAmp = 0;
            *ADC = 0;
            *Ext = 0;
        }
        
		/* check TR point & end of time point */
		 if (*dt <= 0){ 
			if (g_Sig.size() !=0){
				/* calculate signal length */
				SignalLen = Signali-Signalptr;

				/* reset buffer if needed */
				if (PreSignalLen!=SignalLen && SignalLen>0){
					PreSignalLen = SignalLen;
					/* allocate device memory for acquired signal buffer */
					hipFree(d_Sx);
					hipFree(d_Sy);
					hipMalloc( (void**) &d_Sx, SpinMxNum * SignalLen * (*TypeNum) * (*RxCoilNum) * sizeof(float)) ;
					hipMalloc( (void**) &d_Sy, SpinMxNum * SignalLen * (*TypeNum) * (*RxCoilNum) * sizeof(float)) ;
					/* zero signal buffer */
					hipMemset(d_Sx, 0 ,SpinMxNum * SignalLen * (*TypeNum) * (*RxCoilNum) * sizeof(float)); /* only work for 0 */
					hipMemset(d_Sy, 0 ,SpinMxNum * SignalLen * (*TypeNum) * (*RxCoilNum) * sizeof(float)); /* only work for 0 */
					/* set buffer */
					ippsFree(Sxbuffer);
					ippsFree(Sybuffer);
					Sxbuffer = ippsMalloc_32f(SpinMxNum * SignalLen * (*TypeNum) * (*RxCoilNum));
					Sybuffer = ippsMalloc_32f(SpinMxNum * SignalLen * (*TypeNum) * (*RxCoilNum));
				}

				/* avoid shared memory overflow */
				if (g_Sig.size() * sizeof(float) > deviceProp.sharedMemPerBlock){
					SBufferLen = 0;
				}else{
					SBufferLen = g_Sig.size() * sizeof(float);
				}

				/* upload GPU sequence */
				hipMemcpy( d_Sig, 	&g_Sig[0], 	g_Sig.size() * sizeof(float),	hipMemcpyHostToDevice ) ;

				/* call GPU kernel for spin discrete precessing */
				BlochKernelNormalGPU<<< dimGridImg, dimBlockImg, SBufferLen >>>
									((float)*Gyro, d_CS, d_Rho, d_T1, d_T2, d_Mz, d_My, d_Mx,
									d_dB0, d_dWRnd, d_Gzgrid, d_Gygrid, d_Gxgrid, d_TxCoilmg, d_TxCoilpe, d_RxCoilx, d_RxCoily,
									d_Sig, (float)*RxCoilDefault, (float)*TxCoilDefault,
									d_Sx, d_Sy, (float)*rfRef, SignalLen, SBufferLen,
									SpinMxColNum, SpinMxRowNum, SpinMxSliceNum, *SpinNum, *TypeNum, *TxCoilNum, *RxCoilNum, g_Sig.size()/(5+3*(*TxCoilNum)));
				hipDeviceSynchronize(); /* stablize simulation */
				g_Sig.clear();
				Signalptr = Signali;
			}
			
			/* signal acquisition */
			if (SignalLen>0){
				/* get Sx, Sy buffer from GPU */
				hipMemcpy( Sybuffer, d_Sy, SpinMxNum * SignalLen * (*RxCoilNum) * (*TypeNum) * sizeof(float), hipMemcpyDeviceToHost ) ;
				hipMemcpy( Sxbuffer, d_Sx, SpinMxNum * SignalLen * (*RxCoilNum) * (*TypeNum) * sizeof(float), hipMemcpyDeviceToHost ) ;
				
				/* sum MR signal via openMP */
				for (Typei = 0; Typei < *TypeNum; Typei++){
					for (RxCoili = 0; RxCoili < *RxCoilNum; RxCoili++){  /* signal acquisition per Rx coil */
						#pragma omp parallel
						{   
							#pragma omp for private(j, s, p_Sx, p_Sy, buffer) 
							for (j=0; j < SignalLen; j++){
								
								if (j==0){
									*ActiveThreadNum = omp_get_num_threads();
								}
								
								s=Signali-SignalLen+j;
								p_Sx = Sx + (Typei*(*RxCoilNum)*(*SignalNum)+RxCoili*(*SignalNum)+s);
								p_Sy = Sy + (Typei*(*RxCoilNum)*(*SignalNum)+RxCoili*(*SignalNum)+s);
							
								ippsSum_32f(&Sxbuffer[Typei * (SpinMxNum * SignalLen * (*RxCoilNum)) + RxCoili * (SpinMxNum * SignalLen) +  j*SpinMxNum], SpinMxNum, &buffer, ippAlgHintFast);
								*p_Sx = (double)buffer;
								ippsSum_32f(&Sybuffer[Typei * (SpinMxNum * SignalLen * (*RxCoilNum)) + RxCoili * (SpinMxNum * SignalLen) +  j*SpinMxNum], SpinMxNum, &buffer, ippAlgHintFast);
								*p_Sy = (double)buffer;
							
							}
						}
					}       
				}
				
				/* zero signal buffer */
				hipMemset(d_Sx, 0 ,SpinMxNum * SignalLen * (*TypeNum) * (*RxCoilNum) * sizeof(float)); /* only work for 0 */
				hipMemset(d_Sy, 0 ,SpinMxNum * SignalLen * (*TypeNum) * (*RxCoilNum) * sizeof(float)); /* only work for 0 */
			}

			if (*dt < 0){
				(*TRCount)++;
				mexPrintf("TR Counts: %d of %d\n", *TRCount, *TRNum);
			}
        }
    }

    /* free GPU memory */
    hipFree(d_Mz);
    hipFree(d_My);
    hipFree(d_Mx);
    hipFree(d_dWRnd);
    hipFree(d_Rho);
    hipFree(d_T1);
    hipFree(d_T2);
    hipFree(d_Gzgrid);
    hipFree(d_Gygrid);
    hipFree(d_Gxgrid);
    hipFree(d_dB0);
    hipFree(d_TxCoilmg);
    hipFree(d_TxCoilpe);
	hipFree(d_RxCoilx);
    hipFree(d_RxCoily);
    hipFree(d_CS);
    hipFree(d_Sig);
	hipFree(d_Sx);
	hipFree(d_Sy);
	
	/* reset device, may slow down subsequent startup due to initialization */
	// hipDeviceReset();
    
}
